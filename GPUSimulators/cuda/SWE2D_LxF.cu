#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements the classical Lax-Friedrichs scheme
for the shallow water equations, with edge fluxes.

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "common.h"
#include "SWECommon.h"


/**
  * Computes the flux along the x axis for all faces
  */
template <int block_width, int block_height>
__device__ 
void computeFluxF(float Q[3][block_height+2][block_width+2],
                  float F[3][block_height][block_width+1],
                  const float g_, const float dx_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    {
        const int j=ty;
        const int l = j + 1; //Skip ghost cells
        for (int i=tx; i<block_width+1; i+=block_width) {
            const int k = i;
            
            // Q at interface from the right and left
            const float3 Qp = make_float3(Q[0][l][k+1],
                                          Q[1][l][k+1],
                                          Q[2][l][k+1]);
            const float3 Qm = make_float3(Q[0][l][k],
                                          Q[1][l][k],
                                          Q[2][l][k]);
                                       
            // Computed flux
            const float3 flux = LxF_2D_flux(Qm, Qp, g_, dx_, dt_);
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
        }
    }
}


/**
  * Computes the flux along the y axis for all faces
  */ 
template <int block_width, int block_height>
__device__
void computeFluxG(float Q[3][block_height+2][block_width+2],
                  float G[3][block_height+1][block_width],
                  const float g_, const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    for (int j=ty; j<block_height+1; j+=block_height) {
        const int l = j;
        {
            const int i=tx;
            const int k = i + 1; //Skip ghost cells
            
            // Q at interface from the right and left
            // Note that we swap hu and hv
            const float3 Qp = make_float3(Q[0][l+1][k],
                                          Q[2][l+1][k],
                                          Q[1][l+1][k]);
            const float3 Qm = make_float3(Q[0][l][k],
                                          Q[2][l][k],
                                          Q[1][l][k]);

            // Computed flux
            // Note that we swap back
            const float3 flux = LxF_2D_flux(Qm, Qp, g_, dy_, dt_);
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
        }
    }  
}


extern "C" {
__global__ 
void LxFKernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float g_,
        
        int boundary_conditions_,
        
        //Input h^n
        float* h0_ptr_, int h0_pitch_,
        float* hu0_ptr_, int hu0_pitch_,
        float* hv0_ptr_, int hv0_pitch_,
        
        //Output h^{n+1}
        float* h1_ptr_, int h1_pitch_,
        float* hu1_ptr_, int hu1_pitch_,
        float* hv1_ptr_, int hv1_pitch_,
        
       //Output CFL
       float* cfl_,

       //Subarea of internal domain to compute
       int x0=0, int y0=0,
       int x1=0, int y1=0) {

    if(x1 == 0)
        x1 = nx_;

    if(y1 == 0)
        y1 = ny_;
    
    const unsigned int w = BLOCK_WIDTH;
    const unsigned int h = BLOCK_HEIGHT;
    const unsigned int gc_x = 1;
    const unsigned int gc_y = 1;
    const unsigned int vars = 3;
    
    __shared__ float Q[vars][h+2][w+2];
    __shared__ float F[vars][h  ][w+1];
    __shared__ float G[vars][h+1][w  ];
    
    //Read from global memory
    readBlock<w, h, gc_x, gc_y,  1,  1>( h0_ptr_,  h0_pitch_, Q[0], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y, -1,  1>(hu0_ptr_, hu0_pitch_, Q[1], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y,  1, -1>(hv0_ptr_, hv0_pitch_, Q[2], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    
    //Compute fluxes along the x and y axis
    computeFluxF<w, h>(Q, F, g_, dx_, dt_);
    computeFluxG<w, h>(Q, G, g_, dy_, dt_);
    __syncthreads();

    //Evolve for all cells
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int i = tx + 1; //Skip local ghost cells, i.e., +1
    const int j = ty + 1;
    
    Q[0][j][i] += (F[0][ty][tx] - F[0][ty  ][tx+1]) * dt_ / dx_ 
                + (G[0][ty][tx] - G[0][ty+1][tx  ]) * dt_ / dy_;
    Q[1][j][i] += (F[1][ty][tx] - F[1][ty  ][tx+1]) * dt_ / dx_ 
                + (G[1][ty][tx] - G[1][ty+1][tx  ]) * dt_ / dy_;
    Q[2][j][i] += (F[2][ty][tx] - F[2][ty  ][tx+1]) * dt_ / dx_ 
                + (G[2][ty][tx] - G[2][ty+1][tx  ]) * dt_ / dy_;
    __syncthreads();

    //Write to main memory
    writeBlock<w, h, gc_x, gc_y>( h1_ptr_,  h1_pitch_, Q[0], nx_, ny_, 0, 1, x0, y0, x1, y1);
    writeBlock<w, h, gc_x, gc_y>(hu1_ptr_, hu1_pitch_, Q[1], nx_, ny_, 0, 1, x0, y0, x1, y1);
    writeBlock<w, h, gc_x, gc_y>(hv1_ptr_, hv1_pitch_, Q[2], nx_, ny_, 0, 1, x0, y0, x1, y1);
    
    //Compute the CFL for this block
    if (cfl_ != NULL) {
        writeCfl<w, h, gc_x, gc_y, vars>(Q, Q[0], nx_, ny_, dx_, dy_, g_, cfl_);
    }
}

} // extern "C"

