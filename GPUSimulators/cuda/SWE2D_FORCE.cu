#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements the classical Lax-Friedrichs scheme
for the shallow water equations, with edge fluxes.

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "common.h"
#include "SWECommon.h"


/**
  * Computes the flux along the x axis for all faces
  */
__device__ 
void computeFluxF(float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float F[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  const float g_, const float dx_, const float dt_) {
    //Compute fluxes along the x axis
    for (int j=threadIdx.y; j<BLOCK_HEIGHT+2; j+=BLOCK_HEIGHT) {
        for (int i=threadIdx.x; i<BLOCK_WIDTH+1; i+=BLOCK_WIDTH) {
            // Q at interface from the right and left
            const float3 Qp = make_float3(Q[0][j][i+1],
                                          Q[1][j][i+1],
                                          Q[2][j][i+1]);
            const float3 Qm = make_float3(Q[0][j][i],
                                          Q[1][j][i],
                                          Q[2][j][i]);
                                       
            // Computed flux
            const float3 flux = FORCE_1D_flux(Qm, Qp, g_, dx_, dt_);
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
        }
    }
}


/**
  * Computes the flux along the y axis for all faces
  */
__device__ 
void computeFluxG(float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float G[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  const float g_, const float dy_, const float dt_) {
    //Compute fluxes along the y axis
    for (int j=threadIdx.y; j<BLOCK_HEIGHT+1; j+=BLOCK_HEIGHT) {
        for (int i=threadIdx.x; i<BLOCK_WIDTH+2; i+=BLOCK_WIDTH) {
            // Q at interface from the right and left
            // Note that we swap hu and hv
            const float3 Qp = make_float3(Q[0][j+1][i],
                                          Q[2][j+1][i],
                                          Q[1][j+1][i]);
            const float3 Qm = make_float3(Q[0][j][i],
                                          Q[2][j][i],
                                          Q[1][j][i]);

            // Computed flux
            // Note that we swap back
            const float3 flux = FORCE_1D_flux(Qm, Qp, g_, dy_, dt_);
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
        }
    }
}


extern "C" {
__global__ void FORCEKernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float g_,
        
        int boundary_conditions_,
        
        //Input h^n
        float* h0_ptr_, int h0_pitch_,
        float* hu0_ptr_, int hu0_pitch_,
        float* hv0_ptr_, int hv0_pitch_,
        
        //Output h^{n+1}
        float* h1_ptr_, int h1_pitch_,
        float* hu1_ptr_, int hu1_pitch_,
        float* hv1_ptr_, int hv1_pitch_,
        
        //Output CFL
        float* cfl_,

        //Subarea of internal domain to compute
        int x0=0, int y0=0,
        int x1=0, int y1=0) {

    if(x1 == 0)
        x1 = nx_;

    if(y1 == 0)
        y1 = ny_; 

    const unsigned int w = BLOCK_WIDTH;
    const unsigned int h = BLOCK_HEIGHT;
    const unsigned int gc_x = 1;
    const unsigned int gc_y = 1;
    const unsigned int vars = 3;
    
    __shared__ float Q[vars][h+2*gc_y][w+2*gc_x];
    __shared__ float F[vars][h+2*gc_y][w+2*gc_x];
    
    //Read into shared memory
    readBlock<w, h, gc_x, gc_y,  1,  1>( h0_ptr_,  h0_pitch_, Q[0], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y, -1,  1>(hu0_ptr_, hu0_pitch_, Q[1], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y,  1, -1>(hv0_ptr_, hv0_pitch_, Q[2], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    __syncthreads();
    
    //Compute flux along x, and evolve
    computeFluxF(Q, F, g_, dx_, dt_);
    __syncthreads();
    evolveF<w, h, gc_x, gc_y, vars>(Q, F, dx_, dt_);
    __syncthreads();
    
    //Compute flux along y, and evolve
    computeFluxG(Q, F, g_, dy_, dt_);
    __syncthreads();
    evolveG<w, h, gc_x, gc_y, vars>(Q, F, dy_, dt_);
    __syncthreads();
    
    //Write to main memory
    writeBlock<w, h, gc_x, gc_y>( h1_ptr_,  h1_pitch_, Q[0], nx_, ny_, 0, 1, x0, y0, x1, y1);
    writeBlock<w, h, gc_x, gc_y>(hu1_ptr_, hu1_pitch_, Q[1], nx_, ny_, 0, 1, x0, y0, x1, y1);
    writeBlock<w, h, gc_x, gc_y>(hv1_ptr_, hv1_pitch_, Q[2], nx_, ny_, 0, 1, x0, y0, x1, y1);
    
    //Compute the CFL for this block
    if (cfl_ != NULL) {
        writeCfl<w, h, gc_x, gc_y, vars>(Q, F[0], nx_, ny_, dx_, dy_, g_, cfl_);
    }
}

} // extern "C"