#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements the Kurganov-Petrova numerical scheme 
for the shallow water equations, described in 
A. Kurganov & Guergana Petrova
A Second-Order Well-Balanced Positivity Preserving Central-Upwind
Scheme for the Saint-Venant System Communications in Mathematical
Sciences, 5 (2007), 133-160. 

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



#include "common.h"
#include "SWECommon.h"
#include "limiters.h"


template <int w, int h, int gc_x, int gc_y>
__device__
void computeFluxF(float Q[3][h+2*gc_y][w+2*gc_x],
                  float Qx[3][h+2*gc_y][w+2*gc_x],
                  float F[3][h+2*gc_y][w+2*gc_x],
                  const float g_, const float dx_, const float dt_) {
    for (int j=threadIdx.y; j<h+2*gc_y; j+=h) {
        for (int i=threadIdx.x+1; i<w+2*gc_x-2; i+=w) {
            // Reconstruct point values of Q at the left and right hand side 
            // of the cell for both the left (i) and right (i+1) cell 
            const float3 Q_rl = make_float3(Q[0][j][i+1] - 0.5f*Qx[0][j][i+1],
                                            Q[1][j][i+1] - 0.5f*Qx[1][j][i+1],
                                            Q[2][j][i+1] - 0.5f*Qx[2][j][i+1]);
            const float3 Q_rr = make_float3(Q[0][j][i+1] + 0.5f*Qx[0][j][i+1],
                                            Q[1][j][i+1] + 0.5f*Qx[1][j][i+1],
                                            Q[2][j][i+1] + 0.5f*Qx[2][j][i+1]);

            const float3 Q_ll = make_float3(Q[0][j][i] - 0.5f*Qx[0][j][i],
                                            Q[1][j][i] - 0.5f*Qx[1][j][i],
                                            Q[2][j][i] - 0.5f*Qx[2][j][i]);
            const float3 Q_lr = make_float3(Q[0][j][i] + 0.5f*Qx[0][j][i],
                                            Q[1][j][i] + 0.5f*Qx[1][j][i],
                                            Q[2][j][i] + 0.5f*Qx[2][j][i]);
                                    
            //Evolve half a timestep (predictor step)
            const float3 Q_r_bar = Q_rl + dt_/(2.0f*dx_) * (F_func(Q_rl, g_) - F_func(Q_rr, g_));
            const float3 Q_l_bar = Q_lr + dt_/(2.0f*dx_) * (F_func(Q_ll, g_) - F_func(Q_lr, g_));

            // Compute flux based on prediction
            const float3 flux = CentralUpwindFlux(Q_l_bar, Q_r_bar, g_);
            
            //Write to shared memory
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
        }
    }    
}

template <int w, int h, int gc_x, int gc_y>
__device__
void computeFluxG(float Q[3][h+2*gc_y][w+2*gc_x],
                  float Qy[3][h+2*gc_y][w+2*gc_x],
                  float G[3][h+2*gc_y][w+2*gc_x],
                  const float g_, const float dy_, const float dt_) {
    for (int j=threadIdx.y+1; j<h+2*gc_y-2; j+=h) {
        for (int i=threadIdx.x; i<w+2*gc_x; i+=w) {
            // Reconstruct point values of Q at the left and right hand side 
            // of the cell for both the left (i) and right (i+1) cell 
            //NOte that hu and hv are swapped ("transposing" the domain)!
            const float3 Q_rl = make_float3(Q[0][j+1][i] - 0.5f*Qy[0][j+1][i],
                                            Q[2][j+1][i] - 0.5f*Qy[2][j+1][i],
                                            Q[1][j+1][i] - 0.5f*Qy[1][j+1][i]);
            const float3 Q_rr = make_float3(Q[0][j+1][i] + 0.5f*Qy[0][j+1][i],
                                            Q[2][j+1][i] + 0.5f*Qy[2][j+1][i],
                                            Q[1][j+1][i] + 0.5f*Qy[1][j+1][i]);
                                        
            const float3 Q_ll = make_float3(Q[0][j][i] - 0.5f*Qy[0][j][i],
                                            Q[2][j][i] - 0.5f*Qy[2][j][i],
                                            Q[1][j][i] - 0.5f*Qy[1][j][i]);
            const float3 Q_lr = make_float3(Q[0][j][i] + 0.5f*Qy[0][j][i],
                                            Q[2][j][i] + 0.5f*Qy[2][j][i],
                                            Q[1][j][i] + 0.5f*Qy[1][j][i]);
                                     
            //Evolve half a timestep (predictor step)
            const float3 Q_r_bar = Q_rl + dt_/(2.0f*dy_) * (F_func(Q_rl, g_) - F_func(Q_rr, g_));
            const float3 Q_l_bar = Q_lr + dt_/(2.0f*dy_) * (F_func(Q_ll, g_) - F_func(Q_lr, g_));
            
            // Compute flux based on prediction
            const float3 flux = CentralUpwindFlux(Q_l_bar, Q_r_bar, g_);
            
            //Write to shared memory
            //Note that we here swap hu and hv back to the original
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
        }
    }
}




/**
  * This unsplit kernel computes the 2D numerical scheme with a TVD RK2 time integration scheme
  */
extern "C" {
    
    
    
    
__global__ void KP07DimsplitKernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float g_,
        
        float theta_,
        
        int step_,
        int boundary_conditions_,
        
        //Input h^n
        float* h0_ptr_, int h0_pitch_,
        float* hu0_ptr_, int hu0_pitch_,
        float* hv0_ptr_, int hv0_pitch_,
        
        //Output h^{n+1}
        float* h1_ptr_, int h1_pitch_,
        float* hu1_ptr_, int hu1_pitch_,
        float* hv1_ptr_, int hv1_pitch_, 
        
        //Output CFL
        float* cfl_,

        //Subarea of internal domain to compute
        int x0=0, int y0=0,
        int x1=0, int y1=0) {

    if(x1 == 0)
        x1 = nx_;

    if(y1 == 0)
        y1 = ny_;

    const unsigned int w = BLOCK_WIDTH;
    const unsigned int h = BLOCK_HEIGHT;
    const unsigned int gc_x = 2;
    const unsigned int gc_y = 2;
    const unsigned int vars = 3;
        
    //Shared memory variables
    __shared__ float  Q[vars][h+2*gc_y][w+2*gc_x];
    __shared__ float Qx[vars][h+2*gc_y][w+2*gc_x];
    __shared__ float  F[vars][h+2*gc_y][w+2*gc_x];
    
    //Read into shared memory
    readBlock<w, h, gc_x, gc_y,  1,  1>( h0_ptr_,  h0_pitch_, Q[0], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y, -1,  1>(hu0_ptr_, hu0_pitch_, Q[1], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y,  1, -1>(hv0_ptr_, hv0_pitch_, Q[2], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    
    if (step_ == 0) {
        //Along X
        minmodSlopeX<w, h, gc_x, gc_y, vars>(Q, Qx, theta_);
        __syncthreads();
        computeFluxF<w, h, gc_x, gc_y>(Q, Qx, F, g_, dx_, dt_);
        __syncthreads();
        evolveF<w, h, gc_x, gc_y, vars>(Q, F, dx_, dt_);
        __syncthreads();
        
        //Along Y
        minmodSlopeY<w, h, gc_x, gc_y, vars>(Q, Qx, theta_);
        __syncthreads();
        computeFluxG<w, h, gc_x, gc_y>(Q, Qx, F, g_, dy_, dt_);
        __syncthreads();
        evolveG<w, h, gc_x, gc_y, vars>(Q, F, dy_, dt_);
        __syncthreads();
    }
    else {
        //Along Y
        minmodSlopeY<w, h, gc_x, gc_y, vars>(Q, Qx, theta_);
        __syncthreads();
        computeFluxG<w, h, gc_x, gc_y>(Q, Qx, F, g_, dy_, dt_);
        __syncthreads();
        evolveG<w, h, gc_x, gc_y, vars>(Q, F, dy_, dt_);
        __syncthreads();
        
        //Along X
        minmodSlopeX<w, h, gc_x, gc_y, vars>(Q, Qx, theta_);
        __syncthreads();
        computeFluxF<w, h, gc_x, gc_y>(Q, Qx, F, g_, dx_, dt_);
        __syncthreads();
        evolveF<w, h, gc_x, gc_y, vars>(Q, F, dx_, dt_);
        __syncthreads();
    }
    
    // Write to main memory for all internal cells
    writeBlock<w, h, gc_x, gc_y>( h1_ptr_,  h1_pitch_, Q[0], nx_, ny_, 0, 1, x0, y0, x1, y1);
    writeBlock<w, h, gc_x, gc_y>(hu1_ptr_, hu1_pitch_, Q[1], nx_, ny_, 0, 1, x0, y0, x1, y1);
    writeBlock<w, h, gc_x, gc_y>(hv1_ptr_, hv1_pitch_, Q[2], nx_, ny_, 0, 1, x0, y0, x1, y1);
    
    //Compute the CFL for this block
    if (cfl_ != NULL) {
        writeCfl<w, h, gc_x, gc_y, vars>(Q, F[0], nx_, ny_, dx_, dy_, g_, cfl_);
    }
}










} // extern "C"