#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements the Kurganov-Petrova numerical scheme 
for the shallow water equations, described in 
A. Kurganov & Guergana Petrova
A Second-Order Well-Balanced Positivity Preserving Central-Upwind
Scheme for the Saint-Venant System Communications in Mathematical
Sciences, 5 (2007), 133-160. 

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



#include "common.h"
#include "SWECommon.h"
#include "limiters.h"


__device__
void computeFluxF(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qx[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float F[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
                  const float g_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    {
        int j=ty;
        const int l = j + 2; //Skip ghost cells
        for (int i=tx; i<BLOCK_WIDTH+1; i+=BLOCK_WIDTH) {
            const int k = i + 1;
            // Q at interface from the right and left
            const float3 Qp = make_float3(Q[0][l][k+1] - 0.5f*Qx[0][j][i+1],
                                          Q[1][l][k+1] - 0.5f*Qx[1][j][i+1],
                                          Q[2][l][k+1] - 0.5f*Qx[2][j][i+1]);
            const float3 Qm = make_float3(Q[0][l][k  ] + 0.5f*Qx[0][j][i  ],
                                          Q[1][l][k  ] + 0.5f*Qx[1][j][i  ],
                                          Q[2][l][k  ] + 0.5f*Qx[2][j][i  ]);
                                       
            // Computed flux
            const float3 flux = CentralUpwindFlux(Qm, Qp, g_);
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
        }
    }    
}

__device__
void computeFluxG(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qy[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float G[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
                  const float g_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    for (int j=ty; j<BLOCK_HEIGHT+1; j+=BLOCK_HEIGHT) {
        const int l = j + 1;
        {
            int i=tx;
            const int k = i + 2; //Skip ghost cells
            // Q at interface from the right and left
            // Note that we swap hu and hv
            const float3 Qp = make_float3(Q[0][l+1][k] - 0.5f*Qy[0][j+1][i],
                                          Q[2][l+1][k] - 0.5f*Qy[2][j+1][i],
                                          Q[1][l+1][k] - 0.5f*Qy[1][j+1][i]);
            const float3 Qm = make_float3(Q[0][l  ][k] + 0.5f*Qy[0][j  ][i],
                                          Q[2][l  ][k] + 0.5f*Qy[2][j  ][i],
                                          Q[1][l  ][k] + 0.5f*Qy[1][j  ][i]);
                                       
            // Computed flux
            // Note that we swap back
            const float3 flux = CentralUpwindFlux(Qm, Qp, g_);
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
        }
    }
}




__device__ void minmodSlopeX(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qx[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  const float theta_) {
    //Reconstruct slopes along x axis
    for (int p=0; p<3; ++p) {
        {
            const int j = threadIdx.y+2;
            for (int i=threadIdx.x+1; i<BLOCK_WIDTH+3; i+=BLOCK_WIDTH) {
                Qx[p][j-2][i-1] = minmodSlope(Q[p][j][i-1], Q[p][j][i], Q[p][j][i+1], theta_);
            }
        }
    }
}


/**
  * Reconstructs a minmod slope for a whole block along the ordinate
  */
__device__ void minmodSlopeY(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qy[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  const float theta_) {
    //Reconstruct slopes along y axis
    for (int p=0; p<3; ++p) {
        const int i = threadIdx.x + 2;
        for (int j=threadIdx.y+1; j<BLOCK_HEIGHT+3; j+=BLOCK_HEIGHT) {
            {
                Qy[p][j-1][i-2] = minmodSlope(Q[p][j-1][i], Q[p][j][i], Q[p][j+1][i], theta_);
            }
        }
    }
}


/**
  * This unsplit kernel computes the 2D numerical scheme with a TVD RK2 time integration scheme
  */
extern "C" {
__global__ void KP07Kernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float g_,
        
        float theta_,
        
        int step_order_,
        int boundary_conditions_,
        
        //Input h^n
        float* h0_ptr_, int h0_pitch_,
        float* hu0_ptr_, int hu0_pitch_,
        float* hv0_ptr_, int hv0_pitch_,
        
        //Output h^{n+1}
        float* h1_ptr_, int h1_pitch_,
        float* hu1_ptr_, int hu1_pitch_,
        float* hv1_ptr_, int hv1_pitch_,
        
        //Output CFL
        float* cfl_,

        //Subarea of internal domain to compute
        int x0=0, int y0=0,
        int x1=0, int y1=0) {

    if(x1 == 0)
        x1 = nx_;

    if(y1 == 0)
        y1 = ny_;

    const unsigned int w = BLOCK_WIDTH;
    const unsigned int h = BLOCK_HEIGHT;
    const unsigned int gc_x = 2;
    const unsigned int gc_y = 2;
    const unsigned int vars = 3;
        
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    //Index of cell within domain
    const int ti = blockDim.x*blockIdx.x + threadIdx.x + 2; //Skip global ghost cells, i.e., +2
    const int tj = blockDim.y*blockIdx.y + threadIdx.y + 2;
    
    //Shared memory variables
    __shared__ float Q[3][h+4][w+4];
    __shared__ float Qx[3][h+2][w+2];
    __shared__ float Qy[3][h+2][w+2];
    __shared__ float  F[3][h+1][w+1];
    __shared__ float  G[3][h+1][w+1];
    
    
    
    //Read into shared memory
    readBlock<w, h, gc_x, gc_y,  1,  1>( h0_ptr_,  h0_pitch_, Q[0], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y, -1,  1>(hu0_ptr_, hu0_pitch_, Q[1], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y,  1, -1>(hv0_ptr_, hv0_pitch_, Q[2], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    
    
    //Reconstruct slopes along x and axis
    minmodSlopeX(Q, Qx, theta_);
    minmodSlopeY(Q, Qy, theta_);
    __syncthreads();
    
    
    //Compute fluxes along the x and y axis
    computeFluxF(Q, Qx, F, g_);
    computeFluxG(Q, Qy, G, g_);
    __syncthreads();
    
    
    //Sum fluxes and advance in time for all internal cells
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +2
        const int j = ty + 2;
        
        Q[0][j][i] += (F[0][ty][tx] - F[0][ty  ][tx+1]) * dt_ / dx_ 
                    + (G[0][ty][tx] - G[0][ty+1][tx  ]) * dt_ / dy_;
        Q[1][j][i] += (F[1][ty][tx] - F[1][ty  ][tx+1]) * dt_ / dx_ 
                    + (G[1][ty][tx] - G[1][ty+1][tx  ]) * dt_ / dy_;
        Q[2][j][i] += (F[2][ty][tx] - F[2][ty  ][tx+1]) * dt_ / dx_ 
                    + (G[2][ty][tx] - G[2][ty+1][tx  ]) * dt_ / dy_;

        float* const h_row  = (float*) ((char*) h1_ptr_ + h1_pitch_*tj);
        float* const hu_row = (float*) ((char*) hu1_ptr_ + hu1_pitch_*tj);
        float* const hv_row = (float*) ((char*) hv1_ptr_ + hv1_pitch_*tj);

        if (getOrder(step_order_) == 2 && getStep(step_order_) == 1) {
            //Write to main memory
            h_row[ti]  = 0.5f*(h_row[ti]  + Q[0][j][i]);
            hu_row[ti] = 0.5f*(hu_row[ti] + Q[1][j][i]);
            hv_row[ti] = 0.5f*(hv_row[ti] + Q[2][j][i]);
        }
        else {
            h_row[ti]  = Q[0][j][i];
            hu_row[ti] = Q[1][j][i];
            hv_row[ti] = Q[2][j][i];
        }
    }
    
    //Compute the CFL for this block
    if (cfl_ != NULL) {
        writeCfl<w, h, gc_x, gc_y, vars>(Q, Q[0], nx_, ny_, dx_, dy_, g_, cfl_);
    }
}
} //extern "C"